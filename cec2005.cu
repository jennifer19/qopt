
#include <hip/hip_runtime.h>
__constant__ int nreal;
__constant__ int nfunc;

// wszystko ponizsze, co jest stale, chyba powinno byc w constant (wspolne dla wszystkich watkow)
__device__ double C;  // const?
__device__ double global_bias;  // const?
/*
__device__ double *trans_x; // RW (te chyba powinny byc w shared, ale osobne dla kazdego watku,
__device__ double *temp_x1; // RW wiec to chyba powinno byc tablicami 2d)
__device__ double *temp_x2; // RW (jednak w shared to sie nie zmiesci: 50 osobnikow * 50 dim * 8bytes = 20 KB)
__device__ double *temp_x3; // RW (wychodzi na to, ze musi byc w global)
__device__ double *temp_x4; // RW
__device__ double *norm_x;  // RW
__device__ double *basic_f; // nfunc  RW (ok 4 KB w przypadku 50 osobnikow)
__device__ double *weight; // nfunc   RW
__device__ double *norm_f; // nfunc  RW
*/

__device__ double *sigma; // nfunc  // const? (4KB)
__device__ double *lambda; // nfunc  // const? (4KB)
__device__ double *bias; // nfunc  // const? (4KB)
__device__ double **o; // const? (nfunc x nreal) (maks 4 KB)
__device__ double **g; // const? (nreal x nreal) (20KB dla nreal 50)
__device__ double ***l;  // const? (nfunc x nreal x nreal) (maks 200 KB)

__device__ double *l_flat;


// for parallel execution
#define GTID ( blockIdx.y * blockDim.x + threadIdx.x )

__device__ double *g_trans_x; // RW (te chyba powinny byc w shared, ale osobne dla kazdego watku,
__device__ double *g_temp_x1; // RW wiec to chyba powinno byc tablicami 2d)
__device__ double *g_temp_x2; // RW (jednak w shared to sie nie zmiesci: 50 osobnikow * 50 dim * 8bytes = 20 KB)
__device__ double *g_temp_x3; // RW (wychodzi na to, ze musi byc w global)
__device__ double *g_temp_x4; // RW
__device__ double *g_norm_x;  // RW
__device__ double *g_basic_f; // nfunc  RW (ok 4 KB w przypadku 50 osobnikow)
__device__ double *g_weight; // nfunc   RW
__device__ double *g_norm_f; // nfunc  RW

#define trans_x (g_trans_x + nreal * GTID)
#define temp_x1 (g_temp_x1 + nreal * GTID)
#define temp_x2 (g_temp_x2 + nreal * GTID)
#define temp_x3 (g_temp_x3 + nreal * GTID)
#define temp_x4 (g_temp_x4 + nreal * GTID)
#define norm_x  (g_norm_x  + nreal * GTID)
#define basic_f (g_basic_f + nfunc * GTID)
#define weight  (g_weight  + nfunc * GTID)
#define norm_f  (g_norm_f  + nfunc * GTID)


__device__ double calc_sphere (double *x)
{
    int i;
    double res;
    res = 0.0;
    for (i=0; i<nreal; i++)
    {
        res += x[i]*x[i];
    }
    return (res);
}

__device__ double calc_schwefel (double *x)
{
    int i, j;
    double sum1, sum2;
    sum1 = 0.0;
    for (i=0; i<nreal; i++)
    {
        sum2 = 0.0;
        for (j=0; j<=i; j++)
        {
            sum2 += x[j];
        }
        sum1 += sum2*sum2;
    }
    return (sum1);
}

__device__ void transform (double *x, int count)
{
    int i, j;
    for (i=0; i<nreal; i++)
    {
        temp_x1[i] = x[i] - o[count][i];
    }
    for (i=0; i<nreal; i++)
    {
        temp_x2[i] = temp_x1[i]/lambda[count];
    }
    for (j=0; j<nreal; j++)
    {
        temp_x3[j] = 0.0;
        for (i=0; i<nreal; i++)
        {
            temp_x3[j] += g[i][j]*temp_x2[i];
        }
    }
    for (j=0; j<nreal; j++)
    {
        trans_x[j] = 0.0;
        for (i=0; i<nreal; i++)
        {
            // trans_x[j] += l[count][i][j]*temp_x3[i];
            trans_x[j] += l_flat[count * (nreal * nreal) + i * nreal + j] *temp_x3[i];
        }
    }
    return;
}

// F1
__global__ void calc_benchmark_func_f1(double *x, double *res)
{
    transform (x + nreal * GTID, 0);
    basic_f[0] = calc_sphere (trans_x);
    res[GTID] = basic_f[0] + bias[0];
}

// F2
__global__ void calc_benchmark_func_f2(double *x, double *res)
{
    transform (x + nreal * GTID, 0);
    basic_f[0] = calc_schwefel (trans_x);
    res[GTID] = basic_f[0] + bias[0];
}

// F3
__global__ void calc_benchmark_func_f3(double *x, double *res)
{
    int i;
    transform (x + nreal * GTID, 0);
    basic_f[0] = 0.0;
    for (i=0; i<nreal; i++)
    {
        basic_f[0] += trans_x[i]*trans_x[i]*pow(1.0e6,i/(nreal-1.0));
    }
    res[GTID] = basic_f[0] + bias[0];
}

__global__ void f(double *arg, double *result)
{
    result[threadIdx.x] = calc_sphere(arg);
    //result[threadIdx.x] = 1;
}

__global__ void test(double *result, double *o_out, double *g_out, double *l_out) {
    //MEM = *foo;
    //result[0] = sizeof(MEM_t);
    result[0] = l_flat[0];
    result[1] = C;
    result[2] = trans_x[0];
    result[3] = temp_x4[1];
    result[4] = norm_x[0];
    result[5] = norm_f[0];
    result[6] = o[1][0];
    result[7] = bias[0];

    for (int i = 0; i < nfunc; i++) {
        for (int j = 0; j < nreal; j++) {
            o_out[i * nreal + j] = o[i][j];
        }
    }

    for (int i = 0; i < nreal; i++) {
        for (int j = 0; j < nreal; j++) {
            g_out[i * nreal + j] = g[i][j];
        }
    }

    l_out[0] = 13;

    for (int i = 0; i < nfunc * nreal * nreal; i++) {
        l_out[i] = l_flat[i];
    }
}